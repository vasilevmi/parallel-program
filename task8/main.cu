#include "hip/hip_runtime.h"
#include <iostream>
#include <boost/program_options.hpp>
#include <cmath>
#include <memory>
#include <algorithm>
#include <fstream>
#include <iomanip>
#include <chrono>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>

namespace opt = boost::program_options;

template <class ctype>
class Data {
private:
    int len;
    ctype* d_arr;// указатель на массив на GPU

public:
    std::vector<ctype> arr;//сам массив на cpu(хосте)

    Data(int length) : len(length), arr(len), d_arr(nullptr) { //объявление конструктора класса и иниц членов класса . d_arr(nullptr) инициализирует указатель d_arr значением nullptr
        hipError_t err = hipMalloc((void**)&d_arr, len * sizeof(ctype));//err хранит код ошибки
        if (err != hipSuccess) {
            std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(err) << std::endl;//возвращает строку с описанием ошибки
            exit(EXIT_FAILURE);
        }
    }

    ~Data() {
        if (d_arr) {
            hipFree(d_arr);//освобождает память, выделенную для массива на устройстве
        }
    }
    void copyToDevice() {       //данные будут копироваться с хоста на устройство
        hipError_t err = hipMemcpy(d_arr, arr.data(), len * sizeof(ctype), hipMemcpyHostToDevice);//указатель на начало памяти на устройстве (GPU); arr.data() возвращает указатель на начало массива данных вектора.
        if (err != hipSuccess) {
            std::cerr << "CUDA memory copy to device failed: " << hipGetErrorString(err) << std::endl;
            exit(EXIT_FAILURE);
        }
    }
    void copyToHost() {//данные будут копироваться с устройства на хост 
        hipError_t err = hipMemcpy(arr.data(), d_arr, len * sizeof(ctype), hipMemcpyDeviceToHost);// len(длинa массива) * sizeof(ctype)-размер каждого элемента в байтах
        if (err != hipSuccess) {
            std::cerr << "CUDA memory copy to host failed: " << hipGetErrorString(err) << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    ctype* getDevicePointer() { //отдает ссылку на массив на устройстве(гпу) 
        return d_arr;
    }
};

void write_matrix(const double* curmatrix, int N, const std::string& filename) {
    std::ofstream outputFile(filename);
    int fieldWidth = 10;

    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            outputFile << std::setw(fieldWidth) << std::fixed << std::setprecision(4) << curmatrix[i * N + j];
        }
        outputFile << std::endl;
    }

    outputFile.close();
}

double linearInterpolation(double x, double x1, double y1, double x2, double y2) {
    return y1 + ((x - x1) * (y2 - y1) / (x2 - x1));
}



void init(Data<double>& curmatrix, int size) {
    for (int i = 0; i < size; ++i) {
        for (int j = 0; j < size; ++j) {
            curmatrix.arr[i * size + j] = 0;
        }
    }
    curmatrix.arr[0] = 10.0;
    curmatrix.arr[size - 1] = 20.0;
    curmatrix.arr[(size - 1) * size + (size - 1)] = 30.0;
    curmatrix.arr[(size - 1) * size] = 20.0;
    for (int i = 1; i < size - 1; ++i) {
        curmatrix.arr[i * size + 0] = linearInterpolation(i, 0.0, curmatrix.arr[0], size - 1, curmatrix.arr[(size - 1) * size]);
    }
    for (int i = 1; i < size - 1; ++i) {
        curmatrix.arr[0 * size + i] = linearInterpolation(i, 0.0, curmatrix.arr[0], size - 1, curmatrix.arr[size - 1]);
    }
    for (int i = 1; i < size - 1; ++i) {
        curmatrix.arr[(size - 1) * size + i] = linearInterpolation(i, 0.0, curmatrix.arr[(size - 1) * size], size - 1, curmatrix.arr[(size - 1) * size + (size - 1)]);
    }
    for (int i = 1; i < size - 1; ++i) {
        curmatrix.arr[i * size + (size - 1)] = linearInterpolation(i, 0.0, curmatrix.arr[size - 1], size - 1, curmatrix.arr[(size - 1) * size + (size - 1)]);
    }
}

__global__ void iterate(double* curmatrix, double* prevmatrix, int size) { // определяем ядро CUDA с именем iterate. ядро CUDA — это функция, которая выполняется параллельно на устройстве 
    int j = blockIdx.x * blockDim.x + threadIdx.x;//представляет горизонтальную координату
    int i = blockIdx.y * blockDim.y + threadIdx.y;//i — вертикальная координата текущего элемента в массиве.

    if (j == 0 || i == 0 || i >= size - 1 || j >= size - 1) return;//проверяет, находится ли текущий элемент на границе матрицы.Если да то текущий поток не будет обрабатывать данный элемент матрицы, а завершится без выполнения оставшейся части кода

    curmatrix[i * size + j] = 0.25 * (prevmatrix[i * size + j + 1] + prevmatrix[i * size + j - 1] +
        prevmatrix[(i - 1) * size + j] + prevmatrix[(i + 1) * size + j]);
}//Таким образом, это ядро CUDA выполняет одну итерацию вычислений на каждом элементе матрицы

template <unsigned int blockSize>
__global__ void compute_error(double* curmatrix, double* prevmatrix, double* max_error, int size) { //объявление ядра CUDA с именем compute_error
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (j >= size || i >= size) return;//Если координаты элемента находятся за пределами размеров матрицы, то прекращается выполнение текущего потока.

    __shared__ typename hipcub::BlockReduce<double, blockSize>::TempStorage temp_storage;//Общая память доступна только для потоков, выполняющихся внутри одного блока. TempStorage temp_storage для хранения временных данных, необходимых для выполнения операции редукции на уровне блока
    double local_max = 0.0;// максимальное значение ошибки для каждого потока в блоке.


    if (j > 0 && i > 0 && j < size - 1 && i < size - 1) { //вычисляет ошибку для каждого элемента внутри матрицы, за исключением граничных элементов
        int index = i * size + j;
        double error = fabs(curmatrix[index] - prevmatrix[index]);
        local_max = error ;   }

    //  block_max вычисляет максимальное значение ошибки из всех значений в блоке.
    double block_max = hipcub::BlockReduce<double, blockSize>(temp_storage).Reduce(local_max, hipcub::Max());// BlockReduce  используется для выполнения операций редукции на уровне блока ,Max(), которая находит максимальное значение среди всех значений

    if (threadIdx.x == 0 && threadIdx.y == 0) { // является ли текущий поток первым в блоке. !Каждый блок должен записать максимальное значение ошибки только один раз, так как это значение представляет собой максимальную ошибку для всего блока, а не для каждого потока в блоке. 
        int block_index = blockIdx.y * gridDim.x + blockIdx.x;
        max_error[block_index] = block_max;
    }
}

struct CudaFreeDeleter { //для освобождения памяти, выделенной с помощью функции hipMalloc
    void operator()(void* ptr) const {
        hipFree(ptr);
    }
};

struct StreamDeleter {//для уничтожения потока, а затем удаляет указатель.


    void operator()(hipStream_t* stream) {
        hipStreamDestroy(*stream);
        delete stream;
    }
};

struct GraphDeleter { //для уничтожения графа, а затем удаляет указатель.
    void operator()(hipGraph_t* graph) {
        hipGraphDestroy(*graph);
        delete graph;
    }
};

struct GraphExecDeleter { //lля уничтожения исполняемого графа, а затем удаляет указатель.
    void operator()(hipGraphExec_t* graphExec) {
        hipGraphExecDestroy(*graphExec);
        delete graphExec;
    }
};

int main(int argc, char const* argv[]) {
    opt::options_description desc("Arguments");
    desc.add_options()
        ("accuracy", opt::value<double>()->default_value(1e-6), "accuracy")
        ("matr_size", opt::value<int>()->default_value(256), "matrix_size")
        ("num_iter", opt::value<int>()->default_value(1000000), "num_iterations")
        ("help", "help");

    opt::variables_map vm;
    opt::store(opt::parse_command_line(argc, argv, desc), vm);
    opt::notify(vm);// значения опций были успешно разобраны и сохранены в vm
    if (vm.count("help")) {
        std::cout << desc << "\n";
        return 1;
    }

    auto start = std::chrono::high_resolution_clock::now();
    int size = vm["matr_size"].as<int>();
    double accuracy = vm["accuracy"].as<double>();
    int countIter = vm["num_iter"].as<int>();

    Data<double> curmatrix(size * size);
    Data<double> prevmatrix(size * size);

    init(curmatrix, size);// иницилизация значений матрицы 
    init(prevmatrix, size);

    double error;
    error = accuracy + 1;
    int iter = 0;

    dim3 blockDim(32, 32);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x, (size + blockDim.y - 1) / blockDim.y);// определяет размеры сетки. (size + blockDim.x - 1) / blockDim.x: Это вычисляет количество блоков по оси X.(size + blockDim.y - 1) / blockDim.y) по у

    Data<double> d_max_error(gridDim.x * gridDim.y);// массив ошибок размерности сетки gridDim.x * gridDim.y
    Data<double> d_final_max_error(1);
    void* d_temp_storage = nullptr;// указатель, который будет использоваться для выделения памяти на устройстве для временного хранения данных в ходе выполнения операции редукции.
    size_t temp_storage_bytes = 0;//будет содержать количество байтов, необходимых для выделения памяти для временного хранения данных в ходе операции редукции

    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_max_error.getDevicePointer(), d_final_max_error.getDevicePointer(), gridDim.x * gridDim.y);//для выполнения операции редукции на GPU, чтобы найти максимальное значение из данных, хранящихся в d_max_error
    std::unique_ptr<void, CudaFreeDeleter> d_temp_storage_unique;//для автоматического освобождения памяти на устройстве редукции
    hipError_t err = hipMalloc(&d_temp_storage, temp_storage_bytes);// выделяет память на устройстве для временного хранения данных в ходе операции редукции
    if (err != hipSuccess) {
        std::cerr << "CUDA memory allocation failed: " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    d_temp_storage_unique.reset(d_temp_storage);//гарантирует, что память будет автоматически освобождена когда объект d_temp_storage_unique будет уничтожен.

    d_max_error.copyToDevice();
    curmatrix.copyToDevice();
    prevmatrix.copyToDevice();


    //получают указатели на данные, хранящиеся на устройстве.Полезно для передачи этих указателей в качестве аргументов для вызова функций CUDA.
    double* curmatrix_adr = curmatrix.getDevicePointer();
    double* prevmatrix_adr = prevmatrix.getDevicePointer();
    double* d_max_error_adr = d_max_error.getDevicePointer();
    double* d_final_max_error_adr = d_final_max_error.getDevicePointer();

    //Cоздают уникальные указатели для объектов типа hipStream_t, hipGraph_t и hipGraphExec_t
    std::unique_ptr<hipStream_t, StreamDeleter> stream(new hipStream_t);
    std::unique_ptr<hipGraph_t, GraphDeleter> graph(new hipGraph_t);//hipGraph_t представляет собой тип данных из библиотеки CUDA, используемый для представления графа вычислений на GPU
    std::unique_ptr<hipGraphExec_t, GraphExecDeleter> graphExec(new hipGraphExec_t);// для выполнения графа вычислений на GPU

    hipStreamCreate(stream.get());//создает новый поток stream.get() для получения обычного указателя на объект типа hipStream_t
    bool graphCreated = false;

    hipMemset(d_max_error_adr, 0, sizeof(double));// все байты в этой области памяти устанавливаются в 0

    double final_error;


    while (iter < countIter && error > accuracy) { //цикл, который выполняет итерации алгоритма до достижения заданной точности или максимального количества итераций
        if (!graphCreated) {// если граф не создан
            hipStreamBeginCapture(*stream, hipStreamCaptureModeGlobal);//захватываются все операции в потоке, включая ядра CUDA, копирование памяти и другие действия.

            for (int i = 0; i < 999; i++) {
                iterate << <gridDim, blockDim, 0, *stream >> > (curmatrix_adr, prevmatrix_adr, size);//Выполняется серия итераций редукции, представленного в ядре iterate, используя поток stream
                std::swap(prevmatrix_adr, curmatrix_adr);
            }

            iterate << <gridDim, blockDim, 0, *stream >> > (curmatrix_adr, prevmatrix_adr, size);//Это запуск ядра iterate. iterate обрабатывает входные данные (текущее и предыдущее состояния матрицы),
            compute_error<32> << <gridDim, blockDim, 0, *stream >> > (curmatrix_adr, prevmatrix_adr, d_max_error_adr, size);// запуск ядра compute_error. compute_error вычисляет ошибку в результате этих вычислений и записывает ее в d_max_error_adr

            hipStreamEndCapture(*stream, graph.get());//завершает захват операций
            hipGraphInstantiate(graphExec.get(), *graph, nullptr, nullptr, 0);//cоздает  граф вычислений на основе захваченных операций. Указатель на объект графа CUDA, который был захвачен и завершен.

            graphCreated = true;//граф вычислений был создан
        }
        else {
            hipGraphLaunch(*graphExec, *stream);//Граф вычислений запускается для выполнения в потоке stream.
          
            hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_max_error_adr, d_final_max_error_adr, gridDim.x * gridDim.y, *stream);//используется операция редукции для вычисления максимальной ошибки среди всех элементов массива d_max_error_adr на GPU.
            hipMemcpy(&final_error, d_final_max_error_adr, sizeof(double), hipMemcpyDeviceToHost);//Максимальная ошибка копируется с устройства на хост и сохраняется в переменной error
            error = final_error;

            std::cout << "Iteration: " << iter + 1000 << ", Error: " << error << std::endl;

            iter += 1000;
        }
    }

    curmatrix.copyToHost();
    auto end = std::chrono::high_resolution_clock::now();
    auto time_s = std::chrono::duration_cast<std::chrono::milliseconds>(end - start).count();

    std::cout << "time: " << time_s << " error: " << error << " iteration: " << iter << std::endl;

    if (size <= 13) {
        for (size_t i = 0; i < size; i++) {
            for (size_t j = 0; j < size; j++) {
                std::cout << curmatrix.arr[i * size + j] << ' ';
            }
            std::cout << std::endl;
        }
    }

    write_matrix(curmatrix.arr.data(), size, "matrix2.txt");

    return 0;
}
